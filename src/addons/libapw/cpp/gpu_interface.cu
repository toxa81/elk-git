#include <stdio.h>
#include <assert.h>
#include <hipblas.h>
#include <hipblas.h>
#include <magmablas.h>
#include <magma_lapack.h>
#include <magma.h>

#include "typedefs.h"

hipDeviceProp_t& cuda_devprop()
{
    static hipDeviceProp_t devprop;

    return devprop;

}

hipblasHandle_t& cublas_handle()
{
    static hipblasHandle_t handle;
    static bool init = false;

    if (!init)
    {
        if (hipblasCreate(&handle) != HIPBLAS_STATUS_SUCCESS)
        {
            printf("hipblasCreate() failed \n");
            exit(0);
        }
        init = true;
    }
    
    return handle;
}

extern "C" void init_gpu()
{
    int count;
    if (hipGetDeviceCount(&count) != hipSuccess)
    {
        printf("init_gpu: failed to execute hipGetDeviceCount() \n");
        return;
    }

    if (count == 0)
    {
        printf("init_gpu: no avaiable devices\n");
    }

    hipDeviceProp_t devprop;
     
    if (hipGetDeviceProperties(&devprop, 0) != hipSuccess)
    {
        printf("init_gpu: failed to execute hipGetDeviceProperties()\n");
        return;
    }
    
    printf("name                        : %s \n", devprop.name);
    printf("major                       : %i \n", devprop.major);
    printf("minor                       : %i \n", devprop.minor);
    printf("asyncEngineCount            : %i \n", devprop.asyncEngineCount);
    printf("canMapHostMemory            : %i \n", devprop.canMapHostMemory);
    printf("clockRate                   : %i kHz \n", devprop.clockRate);
    printf("concurrentKernels           : %i \n", devprop.concurrentKernels);
    printf("ECCEnabled                  : %i \n", devprop.ECCEnabled);
    printf("l2CacheSize                 : %i kB \n", devprop.l2CacheSize/1024);
    printf("maxGridSize                 : %i %i %i \n", devprop.maxGridSize[0], devprop.maxGridSize[1], devprop.maxGridSize[2]);
    printf("maxThreadsDim               : %i %i %i \n", devprop.maxThreadsDim[0], devprop.maxThreadsDim[1], devprop.maxThreadsDim[2]);
    printf("maxThreadsPerBlock          : %i \n", devprop.maxThreadsPerBlock);
    printf("maxThreadsPerMultiProcessor : %i \n", devprop.maxThreadsPerMultiProcessor);
    printf("memoryBusWidth              : %i bits \n", devprop.memoryBusWidth);
    printf("memoryClockRate             : %i kHz \n", devprop.memoryClockRate);
    printf("memPitch                    : %i \n", devprop.memPitch);
    printf("multiProcessorCount         : %i \n", devprop.multiProcessorCount);
    printf("regsPerBlock                : %i \n", devprop.regsPerBlock);
    printf("sharedMemPerBlock           : %i kB \n", devprop.sharedMemPerBlock/1024);
    printf("totalConstMem               : %i kB \n", devprop.totalConstMem/1024);
    printf("totalGlobalMem              : %i kB \n", devprop.totalGlobalMem/1024);
}

extern "C" void gpu_malloc(void **ptr, int size)
{
    if (hipMalloc(ptr, size) != hipSuccess)
    {
        printf("failed to execute hipMalloc() \n");
        exit(0);
    }
}

extern "C" void gpu_free(void *ptr)
{
    if (hipFree(ptr) != hipSuccess)
    {
        printf("failed to execute hipFree() \n");
        exit(0);
    }
}

extern "C" void gpu_copy_to_device(void *target, void *source, int size)
{
    if (hipMemcpy(target, source, size, hipMemcpyHostToDevice) != hipSuccess)
    {
        printf("failed to execute hipMemcpy(hipMemcpyHostToDevice)\n");
        exit(0);
    }
}

extern "C" void gpu_copy_to_host(void *target, void *source, int size)
{
    if (hipMemcpy(target, source, size, hipMemcpyDeviceToHost) != hipSuccess)
    {
        printf("failed to execute hipMemcpy(hipMemcpyDeviceToHost)\n");
        exit(0);
    }
}

extern "C" void gpu_mem_zero(void *ptr, int size)
{
    if (hipMemset(ptr, 0, size) != hipSuccess)
    {
        printf("failed to execute hipMemset()\n");
        exit(0);
    }
}

extern "C" void gpu_zgemm(int transa, int transb, int32_t m, int32_t n, int32_t k, 
                          complex16 alpha, complex16 *a, int32_t lda, complex16 *b, 
                          int32_t ldb, complex16 beta, complex16 *c, int32_t ldc)
{
    assert(sizeof(hipDoubleComplex) == sizeof(complex16));
    
    const hipblasOperation_t trans[] = {HIPBLAS_OP_N, HIPBLAS_OP_T, HIPBLAS_OP_C};

    if (hipblasZgemm(cublas_handle(), trans[transa], trans[transb], m, n, k, (hipDoubleComplex *)(&alpha), (hipDoubleComplex *)a, lda, 
                    (hipDoubleComplex *)b, ldb, (hipDoubleComplex *)(&beta), (hipDoubleComplex *)c, ldc) != HIPBLAS_STATUS_SUCCESS)
    {
        printf("failed to execute hipblasZgemm() \n");
        exit(0);
    }
}

extern "C" void gpu_zhegvx(int32_t n, int32_t nv, double abstol, void *a, void *b,
                           double *eval, void *z, int32_t ldz)
{
    magma_int_t m1, info;

    magma_int_t nb = magma_get_zhetrd_nb(n);
    magma_int_t lwork = 2 * n * (nb + 1);
    magma_int_t lrwork = 7 * n;
    magma_int_t liwork = 6 * n;
    
    hipDoubleComplex *h_work;
    double *rwork, *w1;
    magma_int_t *iwork, *ifail;
    
    w1 = (double *)malloc(n * sizeof(double));
    h_work = (hipDoubleComplex *)malloc(lwork * sizeof(hipDoubleComplex));
    rwork = (double *)malloc(lrwork * sizeof(double));
    iwork = (magma_int_t *)malloc(liwork * sizeof(magma_int_t));
    ifail = iwork + 5 * n;

    magma_zhegvx(1, 'V', 'I', 'U', n, (hipDoubleComplex *)a, n, (hipDoubleComplex *)b, n, 0.0, 0.0, 1, nv, abstol, 
                 &m1, w1, (hipDoubleComplex *)z, ldz, h_work, lwork, rwork, iwork, ifail, &info);

    memcpy(eval, &w1[0], nv * sizeof(double)); 
    
    free(iwork);
    free(rwork);
    free(w1);
    free(h_work);
}
 
